#include "hip/hip_runtime.h"

#include <iostream>
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "../../array/cuda/utils.h"
#include "../array_scatter.h"
#include <hipcub/hipcub.hpp>   // or equivalently <cub/device/device_radix_sort.cuh>

/**
 * @brief Search for the insertion positions for needle in the hay.
 *
 * The hay is a list of sorted elements and the result is the insertion position
 * of each needle so that the insertion still gives sorted order.
 *
 * It essentially perform binary search to find upper bound for each needle
 * elements.
 *
 * For example:
 * hay = [0, 0, 1, 2, 2]
 * needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 5, 5]
 */
template <typename IdType, typename IndexType>
__global__ void _SortedSearchKernelUpperBound(
    const IdType* hay, int64_t hay_size, const IdType* needles,
    int64_t num_needles, IndexType* pos) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    const IdType ele = needles[tx];
    // binary search
    IdType lo = 0, hi = hay_size;
    while (lo < hi) {
      IdType mid = (lo + hi) >> 1;
      if (hay[mid] <= ele) {
        lo = mid + 1;
      } else {
        hi = mid;
      }
    }
    pos[tx] = lo;
    tx += stride_x;
  }
}

using namespace dgl::runtime;
namespace dgl{
    namespace groot{
        namespace impl {

        // Checks the assigned partition map and scatters the array indices.
        template <typename PIdType, typename IndexType>
        __global__ void scatter_partition_continuous_index_kernel(const PIdType *partition_map,
                                             size_t partition_map_size,
                                             IndexType *index_out, int n_partitions) {
          int tx = blockIdx.x * blockDim.x + threadIdx.x;
          int stride_x = gridDim.x * blockDim.x;
          while (tx < partition_map_size) {
            auto p_id = partition_map[tx];
            assert(p_id < n_partitions);
            index_out[p_id * partition_map_size + tx] = 1;
            tx += stride_x;
          }
        }

        template <typename IDType>
        __inline__ __device__ bool is_selected(const IDType *index, int sz) {
          if (sz == 0) {
            return index[sz] == 1;
          }
          return index[sz] != index[sz - 1];
        }

        // gather_src_idx allows (src[gather_idx] = out
        // scatter_src_idx out[scatter_src_idx] = src
        template <typename IdType>
        __global__ void compute_partition_continuous_index_kernel(size_t source_size, const IdType *index,
            size_t index_size,IdType *gather_src_idx, IdType *scatter_src_idx, IdType * boundary_idx) {
          int tx = blockIdx.x * blockDim.x + threadIdx.x;
          int stride_x = gridDim.x * blockDim.x;
          if(tx == 0){
            boundary_idx[0] = 0;
          }
          while (tx < index_size) {
            if (is_selected(index, tx)) {
              auto value_idx = tx % source_size;
              assert(index[tx] - 1 < source_size);
              gather_src_idx[index[tx] - 1] = value_idx;
              scatter_src_idx[value_idx] = index[tx]-1;

            }
            if((tx + 1) % source_size == 0){
              boundary_idx[(tx + 1)/source_size] = index[tx];
            }
            tx += stride_x;
          }
        }


        // Given a partition map of
        // partition_map = [1,2,3,4,1,2,3,4]
        // returns partition gather index and partition sizes
        // gather_idx = [0,4,1,5,2,6,3,7]
        // scatter_idx = [0,2,4,6,1,3,5,7]
        // gather_idx_sizes = [4,8]
        // out = gather(gather_idx, in)
        // in = scatter(gather_idx, out)
        // in  = gather(scatter_ix, out)
        // out = scatter(gather_idx, in)
        // Naming convention
        // (OP)_in_out_idx, when used in !OP context, we can ensure that in_and_out are flipped
        // such that IndexSelect (F, gather_idx) will result in partition continuos ids
        template <DGLDeviceType XPU, typename IndexType, typename PIdType>
        std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices(IdArray partition_map, \
                                              int num_partitions,hipStream_t stream) {
          // uint8_t nbits = 32;
          CHECK_EQ(partition_map->ndim , 1);
          auto index_data_type = DGLDataTypeTraits<IndexType>::dtype;
          IdArray expanded_index_out = aten::Full<IndexType>(
              0, partition_map->shape[0] * num_partitions, partition_map->ctx);
          size_t partition_map_size = partition_map->shape[0];

          const PIdType *partition_map_idx = partition_map.Ptr<PIdType>();
          IndexType *expanded_idx_ptr = expanded_index_out.Ptr<IndexType>();

          int nt = cuda::FindNumThreads(partition_map_size);
          int nb = (partition_map_size + nt - 1) / nt;
          CUDA_KERNEL_CALL(scatter_partition_continuous_index_kernel, nb, nt, 0, stream,
                           partition_map_idx, partition_map_size, expanded_idx_ptr ,
                           num_partitions);
          size_t workspace_size = 0;
          auto device = runtime::DeviceAPI::Get(partition_map->ctx);
          CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
              nullptr, workspace_size, expanded_idx_ptr , expanded_idx_ptr ,
              partition_map_size * num_partitions, stream));
          void *workspace = device->AllocWorkspace(partition_map->ctx, workspace_size);
          CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
              workspace, workspace_size, expanded_idx_ptr ,expanded_idx_ptr ,
              partition_map_size * num_partitions, stream));
          device->FreeWorkspace(partition_map->ctx, workspace);
          hipStreamSynchronize(stream);


          IdArray gather_idx_in_part_disc_cont =
              IdArray::Empty({partition_map->shape[0]}, index_data_type, partition_map->ctx);
          IdArray scatter_idx_in_part_disc_cont =
              IdArray::Empty({partition_map->shape[0]}, index_data_type, partition_map->ctx);
          IdArray boundary_offsets = IdArray::Empty({num_partitions + 1}, index_data_type, partition_map->ctx);


          IndexType *gather_idx_ptr = gather_idx_in_part_disc_cont.Ptr<IndexType>();
          IndexType *scatter_idx_ptr = scatter_idx_in_part_disc_cont.Ptr<IndexType>();
          IndexType *boundary_offsets_ptr = boundary_offsets.Ptr<IndexType>();
          nt = cuda::FindNumThreads(expanded_index_out->shape[0]);
          nb = (expanded_index_out->shape[0] + nt - 1) / nt;

          CUDA_KERNEL_CALL(compute_partition_continuous_index_kernel, nb, nt, 0, stream,
                               partition_map->shape[0], expanded_idx_ptr,
                                expanded_index_out->shape[0], gather_idx_ptr, scatter_idx_ptr,
                              boundary_offsets_ptr);

          return std::tuple(boundary_offsets, gather_idx_in_part_disc_cont, scatter_idx_in_part_disc_cont);

        }


        template <DGLDeviceType XPU, typename PIdType>
        std::tuple<IdArray,IdArray,IdArray>
        compute_partition_continuous_indices_strawman(IdArray partition_map, \
                                             int num_partitions,hipStream_t stream) {
          typedef PIdType IndexType;
          int num_items = partition_map->shape[0];
          // Declare, allocate, and initialize device-accessible pointers for sorting data
          // e.g., 7
          PIdType  *d_key_buf = partition_map.Ptr<PIdType>();         // e.g., [8, 6, 7, 5, 3, 0, 9]
          IdArray d_key_buf_array_partition = IdArray::Empty({num_items}, partition_map->dtype, partition_map->ctx);
          PIdType  *d_key_alt_buf_partition = d_key_buf_array_partition.Ptr<PIdType>();     // e.g., [        ...        ]
          auto index_data_type = DGLDataTypeTraits<IndexType>::dtype;
          IdArray range = aten::Range(0, num_items, index_data_type.bits, partition_map->ctx);       // e.g., [0, 1, 2, 3, 4, 5, 6]
          IndexType * d_value_buf = range.Ptr<IndexType>();
          IdArray gather_idx_in_part_disc_cont = IdArray::Empty({num_items}, partition_map->dtype, partition_map->ctx);
          IndexType  *d_value_alt_buf = gather_idx_in_part_disc_cont.Ptr<IndexType>(); // e.g., [        ...        ]
//
//          // Create a set of DoubleBuffers to wrap pairs of device pointers
          hipcub::DoubleBuffer<PIdType> d_keys1(d_key_buf, d_key_alt_buf_partition);
          hipcub::DoubleBuffer<IndexType> d_values1(d_value_buf, d_value_alt_buf);
          void     *d_temp_storage = NULL;
          size_t   temp_storage_bytes = 0;

          hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys1, d_values1, num_items, 0, index_data_type.bits, stream);
          auto device = runtime::DeviceAPI::Get(partition_map->ctx);
          d_temp_storage = device->AllocWorkspace(partition_map->ctx,temp_storage_bytes);

      //          // Run sorting operation
          hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys1, d_values1, num_items,  0, index_data_type.bits, stream);
          device->FreeWorkspace(partition_map->ctx, d_temp_storage);

//
          d_key_buf = d_value_alt_buf;
          IdArray d_key_buf_array = IdArray::Empty({num_items}, partition_map->dtype, partition_map->ctx);
          PIdType  *d_key_alt_buf = d_key_buf_array.Ptr<PIdType>();     // e.g., [        ...
          IdArray scatter_idx_in_part_disc_cont = aten::Range(0, num_items, index_data_type.bits, partition_map->ctx);       // e.g., [0, 1, 2, 3, 4, 5, 6]
          d_value_buf = range.Ptr<IndexType>();
          d_value_alt_buf = scatter_idx_in_part_disc_cont.Ptr<IndexType>();
//
          hipcub::DoubleBuffer<PIdType> d_keys2(d_key_buf, d_key_alt_buf);
          hipcub::DoubleBuffer<IndexType> d_values2(d_value_buf, d_value_alt_buf);
//
          hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys2, d_values2, num_items, 0, index_data_type.bits, stream);
          d_temp_storage = device->AllocWorkspace(partition_map->ctx,temp_storage_bytes);

          hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys2, d_values2, num_items, 0, index_data_type.bits, stream);
          device->FreeWorkspace(partition_map->ctx, d_temp_storage);


//        Sorting upper bound kernel compute boudary offsets;
          IdArray needle = IdArray::FromVector(std::vector<IndexType >{0,1,2,3}, partition_map->ctx);
          IdArray boundary_offsets = IdArray::FromVector(std::vector<int64_t>{0,0,0,0,0}, partition_map->ctx);

          const int nt = cuda::FindNumThreads(4);
          const int nb = (4 + nt - 1) / nt;
          CUDA_KERNEL_CALL(
              _SortedSearchKernelUpperBound, nb, nt, 0, stream,  d_key_buf_array_partition.Ptr<IndexType>(),
                  num_items, needle.Ptr<IndexType>(), 4, boundary_offsets.Ptr<int64_t>() + 1);
//          const IdType* hay, int64_t hay_size, const IdType* needles,
//              int64_t num_needles, IdType* pos
          return std::tuple(boundary_offsets, gather_idx_in_part_disc_cont, scatter_idx_in_part_disc_cont);
        }

            template
        std::tuple<IdArray,IdArray,IdArray>
        compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int32_t ,int64_t>(IdArray partition_map, \
                                             int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int64_t ,int64_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);


        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int32_t ,int32_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int64_t ,int32_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices_strawman<DGLDeviceType::kDGLCUDA, int64_t >(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices_strawman<DGLDeviceType::kDGLCUDA, int32_t >(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);



        }
    }
}

