#include "hip/hip_runtime.h"

#include <iostream>
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "../../array/cuda/utils.h"
#include "../array_scatter.h"
using namespace dgl::runtime;
namespace dgl{
    namespace groot{
        namespace impl {

        // Checks the assigned partition map and scatters the array indices.
        template <typename PIdType, typename IndexType>
        __global__ void scatter_partition_continuous_index_kernel(const PIdType *partition_map,
                                             size_t partition_map_size,
                                             IndexType *index_out, int n_partitions) {
          int tx = blockIdx.x * blockDim.x + threadIdx.x;
          int stride_x = gridDim.x * blockDim.x;
          while (tx < partition_map_size) {
            auto p_id = partition_map[tx];
            assert(p_id < n_partitions);
            index_out[p_id * partition_map_size + tx] = 1;
            tx += stride_x;
          }
        }

        template <typename IDType>
        __inline__ __device__ bool is_selected(const IDType *index, int sz) {
          if (sz == 0) {
            return index[sz] == 1;
          }
          return index[sz] != index[sz - 1];
        }

        // gather_src_idx allows (src[gather_idx] = out
        // scatter_src_idx out[scatter_src_idx] = src
        template <typename IdType>
        __global__ void compute_partition_continuous_index_kernel(size_t source_size, const IdType *index,
            size_t index_size,IdType *gather_src_idx, IdType *scatter_src_idx, IdType * boundary_idx) {
          int tx = blockIdx.x * blockDim.x + threadIdx.x;
          int stride_x = gridDim.x * blockDim.x;
          if(tx == 0){
            boundary_idx[0] = 0;
          }
          while (tx < index_size) {
            if (is_selected(index, tx)) {
              auto value_idx = tx % source_size;
              assert(index[tx] - 1 < source_size);
              gather_src_idx[index[tx] - 1] = value_idx;
              scatter_src_idx[value_idx] = index[tx]-1;

            }
            if((tx + 1) % source_size == 0){
              boundary_idx[(tx + 1)/source_size] = index[tx];
            }
            tx += stride_x;
          }
        }


        // Given a partition map of
        // partition_map = [1,2,3,4,1,2,3,4]
        // returns partition gather index and partition sizes
        // gather_idx = [0,5,1,6,2,7,3,8]
        // scatter_idx = [0,2,4,6,1,3,5,7]
        // gather_idx_sizes = [4,8]
        // out = gather(gather_idx, in)
        // in = scatter(gather_idx, out)
        // in  = gather(scatter_ix, out)
        // out = scatter(gather_idx, in)
        // Naming convention
        // (OP)_in_out_idx, when used in !OP context, we can ensure that in_and_out are flipped
        // such that IndexSelect (F, gather_idx) will result in partition continuos ids
        template <DGLDeviceType XPU, typename IndexType, typename PIdType>
        std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices(IdArray partition_map, \
                                              int num_partitions,hipStream_t stream) {
          // uint8_t nbits = 32;
          CHECK_EQ(partition_map->ndim , 1);
          auto index_data_type = DGLDataTypeTraits<IndexType>::dtype;
          IdArray expanded_index_out = aten::Full<IndexType>(
              0, partition_map->shape[0] * num_partitions, partition_map->ctx);
          size_t partition_map_size = partition_map->shape[0];

          const PIdType *partition_map_idx = partition_map.Ptr<PIdType>();
          IndexType *expanded_idx_ptr = expanded_index_out.Ptr<IndexType>();

          int nt = cuda::FindNumThreads(partition_map_size);
          int nb = (partition_map_size + nt - 1) / nt;
          CUDA_KERNEL_CALL(scatter_partition_continuous_index_kernel, nb, nt, 0, stream,
                           partition_map_idx, partition_map_size, expanded_idx_ptr ,
                           num_partitions);
          size_t workspace_size = 0;
          auto device = runtime::DeviceAPI::Get(partition_map->ctx);
          CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
              nullptr, workspace_size, expanded_idx_ptr , expanded_idx_ptr ,
              partition_map_size * num_partitions, stream));
          void *workspace = device->AllocWorkspace(partition_map->ctx, workspace_size);
          CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
              workspace, workspace_size, expanded_idx_ptr ,expanded_idx_ptr ,
              partition_map_size * num_partitions, stream));
          device->FreeWorkspace(partition_map->ctx, workspace);
          hipStreamSynchronize(stream);


          IdArray gather_idx_in_part_disc_cont =
              IdArray::Empty({partition_map->shape[0]}, index_data_type, partition_map->ctx);
          IdArray scatter_idx_in_part_disc_cont =
              IdArray::Empty({partition_map->shape[0]}, index_data_type, partition_map->ctx);
          IdArray boundary_offsets = IdArray::Empty({num_partitions + 1}, index_data_type, partition_map->ctx);


          IndexType *gather_idx_ptr = gather_idx_in_part_disc_cont.Ptr<IndexType>();
          IndexType *scatter_idx_ptr = scatter_idx_in_part_disc_cont.Ptr<IndexType>();
          IndexType *boundary_offsets_ptr = boundary_offsets.Ptr<IndexType>();
          nt = cuda::FindNumThreads(expanded_index_out->shape[0]);
          nb = (expanded_index_out->shape[0] + nt - 1) / nt;

          CUDA_KERNEL_CALL(compute_partition_continuous_index_kernel, nb, nt, 0, stream,
                               partition_map->shape[0], expanded_idx_ptr,
                                expanded_index_out->shape[0], gather_idx_ptr, scatter_idx_ptr,
                              boundary_offsets_ptr);

          return std::tuple(boundary_offsets, gather_idx_in_part_disc_cont, scatter_idx_in_part_disc_cont);

        }


        template
        std::tuple<IdArray,IdArray,IdArray>
        compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int32_t ,int64_t>(IdArray partition_map, \
                                             int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int64_t ,int64_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);


        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int32_t ,int32_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);
        template
            std::tuple<IdArray,IdArray,IdArray>
            compute_partition_continuous_indices<DGLDeviceType::kDGLCUDA, int64_t ,int32_t>(IdArray partition_map, \
                                                                                            int num_partitions,hipStream_t stream);

        }
    }
}

