#include "hip/hip_runtime.h"
//
// Created by juelin on 8/15/23.
//

#include "../../array/cuda/utils.h"
#include "../../runtime/cuda/cuda_common.h"
#include "cuda_index_select.cuh"
#include <assert.h>
#include <dgl/array.h>
#include <dgl/runtime/device_api.h>

namespace dgl {
namespace groot {
namespace impl {

inline std::pair<dim3, dim3>
FindUVAKernelConfig(int feat_len, int feat_width, int feat_bytes = 4,
                    int CUDA_MAX_THREAD_NUM = 1024) {
  CHECK_GE(feat_width, 0);
  if (feat_width == 0)
    return std::make_pair(dim3(feat_len), dim3(1));
  constexpr int warp_size = 32;
  // TODO can we automatically detach this?
  constexpr int max_pcie_requests =
      256; // PCIe3.0 : 256. PCIe4.0: 768, PCIe 5.0: 768?
  // round num threads to the nearest 32 that is no larger than
  // CUDA_MAX_THREAD_NUM
  int num_threads = ((feat_width + warp_size - 1) / warp_size) * warp_size;
  int per_block_request = (num_threads * feat_bytes + 127) / 128;
  ;
  if (num_threads < feat_width) {
    const int scale = (num_threads + feat_width - 1) / feat_width;
    per_block_request *= scale;
  }
  int num_blocks =
      (max_pcie_requests + per_block_request - 1) / per_block_request;
  return std::make_pair(dim3(num_blocks), dim3(num_threads));
}

template <typename DType, typename IdType>
__global__ void IndexSelectSingleKernel(const DType *array, const IdType *index,
                                        const int64_t length,
                                        const int64_t arr_len, DType *out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    assert(index[tx] >= 0 && index[tx] < arr_len);
    out[tx] = array[index[tx]];
    tx += stride_x;
  }
}

template <typename DType, typename IdType>
__global__ void
IndexSelectMultiKernel(const DType *const array, const int64_t num_feat,
                       const IdType *const index, const int64_t length,
                       const int64_t arr_len, DType *const out) {
  int64_t out_row = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (out_row < length) {
    int64_t col = threadIdx.x;
    const int64_t in_row = index[out_row];
    assert(in_row >= 0 && in_row < arr_len);
    while (col < num_feat) {
      out[out_row * num_feat + col] = array[in_row * num_feat + col];
      col += blockDim.x;
    }
    out_row += stride;
  }
}

template <typename DType, typename IdType>
__global__ void
IndexSelectSingleKernel(const DType *array, const IdType *in_index,
                        const IdType *out_index, const int64_t length,
                        const int64_t arr_len, DType *out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    assert(in_index[tx] >= 0 && in_index[tx] < arr_len);
    out[out_index[tx]] = array[in_index[tx]];
    tx += stride_x;
  }
}

template <typename DType, typename IdType>
__global__ void
IndexSelectMultiKernel(const DType *const array, const int64_t num_feat,
                       const IdType *const in_index,
                       const IdType *const out_index, const int64_t length,
                       const int64_t arr_len, DType *const out) {
  int64_t task_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int64_t stride = blockDim.y * gridDim.x;

  while (task_id < length) {
    int64_t col = threadIdx.x;
    const int64_t in_row = in_index[task_id];
    const int64_t out_row = out_index[task_id];
    assert(in_row >= 0 && in_row < arr_len);
    while (col < num_feat) {
      out[out_row * num_feat + col] = array[in_row * num_feat + col];
      col += blockDim.x;
    }
    task_id += stride;
  }
}

template <typename DType, typename IdType>
__global__ void IndexScatterSingleKernel(const DType *array,
                                         const IdType *index,
                                         const int64_t length,
                                         const int64_t arr_len, DType *out) {
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    assert(index[tx] >= 0 && index[tx] < arr_len);
    out[index[tx]] = array[tx];
    tx += stride_x;
  }
}

template <typename DType, typename IdType>
__global__ void
IndexScatterMultiKernel(const DType *const array, const int64_t num_feat,
                        const IdType *const index, const int64_t length,
                        const int64_t arr_len, DType *const out) {
  int64_t in_row = blockIdx.x * blockDim.y + threadIdx.y;

  const int64_t stride = blockDim.y * gridDim.x;

  while (in_row < length) {
    int64_t col = threadIdx.x;
    const int64_t out_row = index[in_row];
    assert(out_row >= 0 && out_row < arr_len);
    while (col < num_feat) {
      out[out_row * num_feat + col] = array[in_row * num_feat + col];
      col += blockDim.x;
    }
    in_row += stride;
  }
}

} // namespace impl
template <typename DType, typename IdType>
NDArray _IndexSelect(NDArray array, IdArray index, hipStream_t stream) {
  bool is_pinned = array.IsPinned();
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  int64_t num_feat = 1;
  std::vector<int64_t> shape{len};
  for (int d = 1; d < array->ndim; ++d) {
    num_feat *= array->shape[d];
    shape.emplace_back(array->shape[d]);
  }

  // use index->ctx for pinned array
  NDArray ret = NDArray::Empty(shape, array->dtype, index->ctx);
  if (len == 0 || arr_len * num_feat == 0)
    return ret;
  DType *ret_data = static_cast<DType *>(ret->data);

  const DType *array_data = static_cast<DType *>(cuda::GetDevicePointer(array));
  const IdType *idx_data = static_cast<IdType *>(index->data);
  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(impl::IndexSelectSingleKernel, nb, nt, 0, stream,
                     array_data, idx_data, len, arr_len, ret_data);
  } else if (!is_pinned) {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, idx_data, len, arr_len, ret_data);
  } else {
    // array is pinned
    const auto config =
        impl::FindUVAKernelConfig(len, num_feat, array->dtype.bits / 8);
    const auto grid = config.first;
    const auto block = config.second;
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, idx_data, len, arr_len, ret_data);
  }
  return ret;
}

template <typename DType, typename IdType>
void _IndexSelect(NDArray array, IdArray index, NDArray &ret,
                  hipStream_t stream) {
  bool is_pinned = array.IsPinned();
  CHECK_EQ(array->ndim , ret->ndim);
  const int64_t arr_len = array->shape[0];
  const int64_t len = index->shape[0];
  int64_t num_feat = 1;
  //            std::vector<int64_t> shape{len};
  for (int d = 1; d < array->ndim; ++d) {
    num_feat *= array->shape[d];
    //                shape.emplace_back(array->shape[d]);
  }

  ret->shape[1] = num_feat;
  ret->shape[0] = len;
  // use index->ctx for pinned array
  if (len == 0 || arr_len * num_feat == 0)
    return;
  DType *ret_data = static_cast<DType *>(ret->data);

  const DType *array_data = static_cast<DType *>(cuda::GetDevicePointer(array));
  const IdType *idx_data = static_cast<IdType *>(index->data);
  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(impl::IndexSelectSingleKernel, nb, nt, 0, stream,
                     array_data, idx_data, len, arr_len, ret_data);
  } else if (!is_pinned) {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, idx_data, len, arr_len, ret_data);
  } else {
    // array is pinned
    const auto config =
        impl::FindUVAKernelConfig(len, num_feat, array->dtype.bits / 8);
    const auto grid = config.first;
    const auto block = config.second;
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, idx_data, len, arr_len, ret_data);
  }
}

template <typename DType, typename IdType>
void _IndexSelect(NDArray array, IdArray in_index, IdArray out_index,
                  NDArray &ret, hipStream_t stream) {
  bool is_pinned = array.IsPinned();
  const int64_t arr_len = array->shape[0];
  const int64_t len = in_index->shape[0];
  int64_t num_feat = 1;
  //            std::vector<int64_t> shape{len};
  for (int d = 1; d < array->ndim; ++d) {
    num_feat *= array->shape[d];
    //                shape.emplace_back(array->shape[d]);
  }

  ret->shape[1] = num_feat;
  ret->shape[0] = len;
  // use in_index->ctx for pinned array
  if (len == 0 || arr_len * num_feat == 0)
    return;
  DType *ret_data = static_cast<DType *>(ret->data);

  const DType *array_data = static_cast<DType *>(cuda::GetDevicePointer(array));
  const IdType *in_index_data = static_cast<IdType *>(in_index->data);
  const IdType *out_index_data = static_cast<IdType *>(out_index->data);

  if (num_feat == 1) {
    const int nt = cuda::FindNumThreads(len);
    const int nb = (len + nt - 1) / nt;
    CUDA_KERNEL_CALL(impl::IndexSelectSingleKernel, nb, nt, 0, stream,
                     array_data, in_index_data, out_index_data, len, arr_len,
                     ret_data);
  } else if (!is_pinned) {
    dim3 block(256, 1);
    while (static_cast<int64_t>(block.x) >= 2 * num_feat) {
      block.x /= 2;
      block.y *= 2;
    }
    const dim3 grid((len + block.y - 1) / block.y);
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, in_index_data, out_index_data, len,
                     arr_len, ret_data);
  } else {
    // array is pinned
    const auto config =
        impl::FindUVAKernelConfig(len, num_feat, array->dtype.bits / 8);
    const auto grid = config.first;
    const auto block = config.second;
    CUDA_KERNEL_CALL(impl::IndexSelectMultiKernel, grid, block, 0, stream,
                     array_data, num_feat, in_index_data, out_index_data, len,
                     arr_len, ret_data);
  }
}

template NDArray _IndexSelect<int8_t, int32_t>(NDArray, IdArray,
                                               hipStream_t stream);

template NDArray _IndexSelect<int8_t, int64_t>(NDArray, IdArray,
                                               hipStream_t stream);

template NDArray _IndexSelect<int16_t, int32_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<int16_t, int64_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<int32_t, int32_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<int32_t, int64_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<int64_t, int32_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<int64_t, int64_t>(NDArray, IdArray,
                                                hipStream_t stream);

template NDArray _IndexSelect<__half, int32_t>(NDArray, IdArray,
                                               hipStream_t stream);

template NDArray _IndexSelect<__half, int64_t>(NDArray, IdArray,
                                               hipStream_t stream);

#if BF16_ENABLED

template NDArray _IndexSelect<__hip_bfloat16, int32_t>(NDArray, IdArray,
                                                      hipStream_t stream);

template NDArray _IndexSelect<__hip_bfloat16, int64_t>(NDArray, IdArray,
                                                      hipStream_t stream);

#endif // BF16_ENABLED

template NDArray _IndexSelect<float, int32_t>(NDArray, IdArray,
                                              hipStream_t stream);

template NDArray _IndexSelect<float, int64_t>(NDArray, IdArray,
                                              hipStream_t stream);

template NDArray _IndexSelect<double, int32_t>(NDArray, IdArray,
                                               hipStream_t stream);

template NDArray _IndexSelect<double, int64_t>(NDArray, IdArray,
                                               hipStream_t stream);

// with buffer

template void _IndexSelect<int8_t, int32_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

template void _IndexSelect<int8_t, int64_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

template void _IndexSelect<int16_t, int32_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<int16_t, int64_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<int32_t, int32_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<int32_t, int64_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<int64_t, int32_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<int64_t, int64_t>(NDArray, IdArray, NDArray &,
                                             hipStream_t stream);

template void _IndexSelect<__half, int32_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

template void _IndexSelect<__half, int64_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

#if BF16_ENABLED

template void _IndexSelect<__hip_bfloat16, int32_t>(NDArray, IdArray, NDArray &,
                                                   hipStream_t stream);

template void _IndexSelect<__hip_bfloat16, int64_t>(NDArray, IdArray, NDArray &,
                                                   hipStream_t stream);

#endif // BF16_ENABLED

template void _IndexSelect<float, int32_t>(NDArray, IdArray, NDArray &,
                                           hipStream_t stream);

template void _IndexSelect<float, int64_t>(NDArray, IdArray, NDArray &,
                                           hipStream_t stream);

template void _IndexSelect<double, int32_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

template void _IndexSelect<double, int64_t>(NDArray, IdArray, NDArray &,
                                            hipStream_t stream);

// with buff and out index

template void _IndexSelect<int8_t, int32_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

template void _IndexSelect<int8_t, int64_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

template void _IndexSelect<int16_t, int32_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<int16_t, int64_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<int32_t, int32_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<int32_t, int64_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<int64_t, int32_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<int64_t, int64_t>(NDArray, IdArray, IdArray,
                                             NDArray &, hipStream_t stream);

template void _IndexSelect<__half, int32_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

template void _IndexSelect<__half, int64_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

#if BF16_ENABLED

template void _IndexSelect<__hip_bfloat16, int32_t>(NDArray, IdArray, IdArray,
                                                   NDArray &,
                                                   hipStream_t stream);

template void _IndexSelect<__hip_bfloat16, int64_t>(NDArray, IdArray, IdArray,
                                                   NDArray &,
                                                   hipStream_t stream);

#endif // BF16_ENABLED

template void _IndexSelect<float, int32_t>(NDArray, IdArray, IdArray, NDArray &,
                                           hipStream_t stream);

template void _IndexSelect<float, int64_t>(NDArray, IdArray, IdArray, NDArray &,
                                           hipStream_t stream);

template void _IndexSelect<double, int32_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

template void _IndexSelect<double, int64_t>(NDArray, IdArray, IdArray,
                                            NDArray &, hipStream_t stream);

NDArray IndexSelect(NDArray array, IdArray index, hipStream_t stream) {
  std::string header = "IndexSelect";
  if (index.NumElements() <= 8192) {
    header += "Label";
  } else {
    if (array.IsPinned()) {
      header += "FeatureUVA";
    } else {
      header += "FeatureGPU";
    }
  }
  nvtx3::scoped_range index_select{header};
  ATEN_DTYPE_BITS_ONLY_SWITCH(array->dtype, DType, "values", {
    ATEN_ID_TYPE_SWITCH(index->dtype, IdType, {
      return _IndexSelect<DType, IdType>(array, index, stream);
    });
  });
};

void IndexSelect(NDArray array, IdArray index, NDArray &out_buff,
                 hipStream_t stream) {
  std::string header = "IndexSelect";
  if (index.NumElements() <= 8192) {
    header += "Label";
  } else {
    if (array.IsPinned()) {
      header += "FeatureUVA";
    } else {
      header += "FeatureGPU";
    }
  }
  nvtx3::scoped_range index_select{header};
  ATEN_DTYPE_BITS_ONLY_SWITCH(array->dtype, DType, "values", {
    ATEN_ID_TYPE_SWITCH(index->dtype, IdType, {
      return _IndexSelect<DType, IdType>(array, index, out_buff, stream);
    });
  });
};

void IndexSelect(NDArray array, IdArray in_index, IdArray out_index,
                 NDArray &out_buff, hipStream_t stream) {
  std::string header = "IndexSelect";
  CHECK_EQ(in_index->dtype, out_index->dtype);
  CHECK_EQ(in_index->shape[0], out_index->shape[0])
      << "in_index and out_index must have the same shape";
  if (in_index.NumElements() <= 8192) {
    header += "Label";
  } else {
    if (array.IsPinned()) {
      header += "FeatureUVA";
    } else {
      header += "FeatureGPU";
    }
  }
  nvtx3::scoped_range index_select{header};
  ATEN_DTYPE_BITS_ONLY_SWITCH(array->dtype, DType, "values", {
    ATEN_ID_TYPE_SWITCH(in_index->dtype, IdType, {
      return _IndexSelect<DType, IdType>(array, in_index, out_index, out_buff,
                                         stream);
    });
  });
};
} // namespace groot
} // namespace dgl