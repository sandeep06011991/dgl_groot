#include "hip/hip_runtime.h"
#include "array_scatter.h"
#include <iostream>
#include <dgl/array.h>
#include "../../runtime/cuda/cuda_common.h"
#include "../../array/cuda/utils.h"
// #include "../array_scatter.h"
#include <iostream>
using namespace dgl::runtime;

namespace dgl{
    namespace groot{
        namespace impl{

template<typename IdType>
__global__
void scatter_index_kernel(
    const IdType * partition_map, size_t partition_map_size, \
         IdType * index_out, int n_partitions){
        int tx = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = gridDim.x * blockDim.x;
        while (tx < partition_map_size) {
            auto p_id = partition_map[tx];
            assert(p_id < n_partitions);
            index_out[p_id * partition_map_size + tx] = 1;
            tx += stride_x;
    }
}

template<typename IdType>
__global__
void get_boundary_offsets_kernel(
    const IdType * index_cum, size_t num_partitions, size_t partition_map_size, \
         IdType * index_out){
        int tx = blockIdx.x * blockDim.x + threadIdx.x;
        int stride_x = gridDim.x * blockDim.x;
        while (tx < num_partitions) {
            index_out[tx] = index_cum[partition_map_size -1 + tx * partition_map_size];
            tx += stride_x;
    }
}


template<typename IdType>
IdArray scatter_index_local(IdArray partition_map,int num_partitions, uint8_t nbits){
    // uint8_t nbits = 32;
    IdArray index_out = aten::Full(0, partition_map->shape[0] * num_partitions, nbits, \
            partition_map->ctx);
    size_t partition_map_size = partition_map->shape[0];
    const IdType* partition_map_idx = partition_map.Ptr<IdType>();
    IdType* index_out_idx = index_out.Ptr<IdType>();

    hipStream_t stream = runtime::getCurrentCUDAStream();
    const int nt = cuda::FindNumThreads(partition_map_size);
    const int nb = (partition_map_size + nt - 1) / nt;
    CUDA_KERNEL_CALL(scatter_index_kernel, nb, nt, 0, stream,\
        partition_map_idx, partition_map_size, index_out_idx, num_partitions);
    size_t workspace_size = 0;
    auto device = runtime::DeviceAPI::Get(index_out->ctx);
    CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
        nullptr, workspace_size,\
                index_out_idx, index_out_idx,\
                    partition_map_size * num_partitions, stream));
    void* workspace = device->AllocWorkspace(index_out->ctx, workspace_size);
    CUDA_CALL(hipcub::DeviceScan::InclusiveSum(
        workspace , workspace_size,\
                index_out_idx, index_out_idx,\
                    partition_map_size * num_partitions, stream));
    hipStreamSynchronize(stream);
    return index_out;
}


    


template IdArray scatter_index_local<int32_t>(IdArray ,int, uint8_t);
template IdArray scatter_index_local<int64_t>(IdArray ,int, uint8_t);

IdArray scatter_index(IdArray partition_map,int num_partitions){
    IdArray ret;
    // Todo Switching here is incorrect 
    if(partition_map->dtype.bits == 32){
        ret = scatter_index_local<int>(partition_map, num_partitions, 32);
    }
    if(partition_map->dtype.bits == 64){
        ret = scatter_index_local<long>(partition_map, num_partitions, 64);
    }
   
    return ret;
}   
template<DGLDeviceType XPU, typename IdType>
IdArray  getBoundaryOffsetsLocal(IdArray index_cum_sums,int num_partitions){    
    IdArray index_out = aten::Full(0,  num_partitions, index_cum_sums->dtype.bits, \
            index_cum_sums->ctx);

    const IdType* index_cum_sums_idx = index_cum_sums.Ptr<IdType>();
    IdType* index_out_idx = index_out.Ptr<IdType>();
    hipStream_t stream = runtime::getCurrentCUDAStream();
    const int nt = cuda::FindNumThreads(num_partitions);
    const int nb = (num_partitions + nt - 1) / nt;
    CUDA_KERNEL_CALL(get_boundary_offsets_kernel, nb, nt, 0, stream,\
        index_cum_sums_idx, num_partitions, index_cum_sums->shape[0]/num_partitions, index_out_idx);

    return index_out.CopyTo(DGLContext{kDGLCPU, 0});  
}

    template IdArray getBoundaryOffsetsLocal<kDGLCUDA, int32_t>(IdArray ,int );
    template IdArray getBoundaryOffsetsLocal<kDGLCUDA, int64_t>(IdArray ,int );
    
        }
    }
}